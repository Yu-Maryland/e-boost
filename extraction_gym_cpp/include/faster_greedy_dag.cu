#include "hip/hip_runtime.h"
// cuda_extractor.cu
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// 这里用简单的常量表示“无穷大”cost
#define INFINITY_COST 1e9f

// 假设每个节点的 cost 是一个浮点数，
// 同时我们将原来的 HashMap cost_set 化简为：
//   每个节点仅保存一个最终的总 cost 和对应的 choice（这里用整型 id 表示）。
struct CostSet {
    float total;
    int   choice;
};

// 假设节点数据（扁平化后的 egraph 节点）
// 注意：这里仅给出一个简化示例，实际数据结构可能要包含更多信息
struct Node {
    int id;             // 节点 id
    int class_id;       // 节点所属的类 id
    float cost;         // 自身 cost
    int numChildren;    // 子节点数量
    int* children;      // 指向子节点 id 的数组（下标对应全局节点数组）
};

// 假设所有节点、costs 等数据已经保存在设备内存中
// 例如：d_nodes 数组、d_costs 数组、以及表示待处理节点索引的队列 d_pending_queue
// 以及每轮新待处理节点数量（用设备内存中的一个 int 保存）
  
// 一个示例 CUDA 内核，每个线程处理队列中的一个节点
__global__ void processNodesKernel(Node* d_nodes,
                                   int    numNodes,
                                   CostSet* d_costs,
                                   int* d_pending_queue, int pendingCount,
                                   int* d_new_queue, int* d_new_count)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= pendingCount) return;
    
    // 取出待处理节点的下标
    int nodeIndex = d_pending_queue[idx];
    Node node = d_nodes[nodeIndex];
    
    // 判断所有子节点是否都已计算出 cost
    bool ready = true;
    for (int i = 0; i < node.numChildren; i++) {
        int childId = node.children[i];
        // 假设：如果子节点的 cost 为 INFINITY_COST，则表示还未更新完成
        if (d_costs[childId].total >= INFINITY_COST) {
            ready = false;
            break;
        }
    }
    if (!ready) return; // 还有子节点未处理，不更新该节点

    // 计算当前节点的新的 cost_set
    // 这里仅做个简单示例：假设新 cost = 自身 cost + 所有子 cost 之和
    float newCost = node.cost;
    for (int i = 0; i < node.numChildren; i++) {
        int childId = node.children[i];
        newCost += d_costs[childId].total;
    }
    
    // 取出原有 cost
    float prevCost = d_costs[nodeIndex].total;
    
    // 如果新 cost 更低，则更新该节点的 cost 并把“父节点”加入新的 pending queue
    if (newCost < prevCost) {
        d_costs[nodeIndex].total = newCost;
        d_costs[nodeIndex].choice = node.id; // 这里简单地将自身 id 作为 choice
        
        // TODO：如果需要更新父节点，则将父节点 id 写入新队列
        // 这里假设每个节点在内存中预先保存有指向父节点的数组（此处省略）
        // 为了示例，我们假设每个节点都有一个唯一的父节点（实际情况可能不止一个）
        // 举例：假设 node.parent 给出父节点的下标（如果有的话，不存在则为 -1）
        // 这里就不展开实现，实际情况中你需要预先构造好父节点列表。
        
        // 举例：如果有父节点，则原子写入到 d_new_queue 中：
        // int parentId = node.parent;
        // if (parentId >= 0) {
        //     int pos = atomicAdd(d_new_count, 1);
        //     d_new_queue[pos] = parentId;
        // }
    }
}

// 主机侧伪代码：调度内核，多轮迭代直到 pending queue 为空
int main() {
    // 1. 初始化、分配并拷贝数据到设备内存
    //    包括：d_nodes（节点数组）、d_costs（每个节点的 cost_set 数组）
    //         d_pending_queue（初始待处理节点下标队列）等。
    //
    // 此处省略具体数据加载的代码，假设你已经根据 egraph 结构完成了扁平化，
    // 并将所有节点信息存入设备内存数组 d_nodes，
    // 同时 d_costs 数组中初始值：叶节点 cost 为自身 cost，其余节点 cost 设置为 INFINITY_COST。
    
    // 2. 分配 pending 队列（两个队列用于交替调度）以及一个设备侧的 int 用于记录新队列长度
    int pendingQueueSize = /* 合适的大小 */;
    int *d_pending_queue, *d_new_queue, *d_new_count;
    hipMalloc(&d_pending_queue, pendingQueueSize * sizeof(int));
    hipMalloc(&d_new_queue, pendingQueueSize * sizeof(int));
    hipMalloc(&d_new_count, sizeof(int));
    
    // 3. 将初始待处理节点 id 拷贝到 d_pending_queue，并设置 pendingCount
    int pendingCount = /* 初始队列节点数量 */;
    
    // 4. 循环调度内核，直到 pending 队列为空
    while (pendingCount > 0) {
        // 将 d_new_count 清零
        hipMemset(d_new_count, 0, sizeof(int));
        
        int blockSize = 256;
        int gridSize = (pendingCount + blockSize - 1) / blockSize;
        processNodesKernel<<<gridSize, blockSize>>>(/* device pointers */ 
                                                    d_nodes, /* numNodes */  /*...*/, d_costs,
                                                    d_pending_queue, pendingCount,
                                                    d_new_queue, d_new_count);
        hipDeviceSynchronize();
        
        // 将 d_new_count 拷贝到 host 以获得新队列大小
        int newCount = 0;
        hipMemcpy(&newCount, d_new_count, sizeof(int), hipMemcpyDeviceToHost);
        
        // 交换 d_pending_queue 和 d_new_queue 指针，准备下一轮迭代
        int* tmp = d_pending_queue;
        d_pending_queue = d_new_queue;
        d_new_queue = tmp;
        pendingCount = newCount;
    }
    
    // 5. 处理完毕后，将最终的 cost_set 结果从 d_costs 拷贝到主机进行后续处理
    // 6. 清理设备内存
    hipFree(d_nodes);
    hipFree(d_costs);
    hipFree(d_pending_queue);
    hipFree(d_new_queue);
    hipFree(d_new_count);
    
    return 0;
}
